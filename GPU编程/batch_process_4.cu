#include "hip/hip_runtime.h"
// 这是程序二的模板程序，我们已经准备好了加载数据集和加载程序一模型参数的部分，请实现CUDA的深度学习推理过程，请严格保持输出格式输出
// 编译的命令为：nvcc test.cu -o test -Xcompiler "-O3 -std=c++14" -gencode arch=compute_50,code=sm_50 -gencode arch=compute_52,code=sm_52 -gencode arch=compute_53,code=sm_53 -gencode arch=compute_60,code=sm_60 -gencode arch=compute_61,code=sm_61 -gencode arch=compute_62,code=sm_62 -gencode arch=compute_70,code=sm_70 -lhdf5 -lhdf5_cpp
// 编译的命令为：
// nvcc batch_process.cu -o batch_process -Xcompiler "-O3 -std=c++14" -gencode arch=compute_50,code=sm_50 -gencode arch=compute_52,code=sm_52 -gencode arch=compute_53,code=sm_53 -gencode arch=compute_60,code=sm_60 -gencode arch=compute_61,code=sm_61 -gencode arch=compute_62,code=sm_62 -gencode arch=compute_70,code=sm_70 -lhdf5 -lhdf5_cpp

#include <iostream>
#include <vector>
#include <chrono>
#include <iomanip>
#include <string>
#include <fstream>
#include <map>
#include <dirent.h>
#include <cstring>
#include <hdf5/serial/H5Cpp.h>
using namespace std;
__global__ void conv1d_norm_relu_begin(float * output,  int a_row, int a_col, int b_col, float * weight, int start_index, int end_index, float ** input, float * bias, float * bn_weight, float * bn_bias, float * bn_running_mean, float * bn_running_var);
__global__ void conv1d_norm_relu_batch(float * output,  int a_row, int a_col, int b_col, float * weight, int current_batch_size,float * input, float * bias, float * bn_weight, float * bn_bias, float * bn_running_mean, float * bn_running_var);
__global__ void conv1d_norm_relu_copy_T_batch(int current_batch_size,float * output,  float * output_T, int a_row, int a_col, int b_col, float * weight, float * input, float * bias, float * bn_weight, float * bn_bias, float * bn_running_mean, float * bn_running_var);
__global__ void conv1d_4096_add_64_64_batch(int current_batch_size,float * output,  int a_row, int a_col, int b_col, float * weight, float * input, float * bias);
__global__ void conv1d_22500_64_64_trans_result_batch(    int current_batch_size,    float * output, int a_row, int a_col, int b_col,     float * input_a, float * input_b);
__global__ void conv1d_log_softmax_ans_batch(int current_batch_size, float * output, int a_row, int a_col, float * weight, float * input, float * bias);
__global__ void conv1d_norm_batch(    int current_batch_size,    float * output,  int a_row, int a_col, int b_col, float * weight, float * input, float * bias, float * bn_weight, float * bn_bias, float * bn_running_mean, float * bn_running_var);
__global__ void max_matrix_batch(float * output, int current_batch_size, float * input, int row, int col);
__global__ void conv1d_9_batch(float * output, float * weight, int current_batch_size, float * input, float * bias);
__global__ void matrix_array_9_batch(int start_index, int end_index, float * output, float ** a, float * b, int a_row, int a_col, int b_col);
__global__ void get_max_ans_batch(  int current_batch_size, float * device_10_1_BATCH,float *device_max_ans_BATCH     );
void print_device_array( float * array, int row, int col, int batch, bool from22400);
void print_device_float(float *num);

#define print_int(x) printf("%d\n", (int)(x))
#define print_float(x) printf("%f\n", (float)(x))
#define forloop(i, start, end) for (int i = (start); i <= (end); ++i)


/****************************************************************************************
 * 读取模型参数
 ****************************************************************************************/
// 获取目录中的所有 .txt 文件
std::vector<std::string> get_files_in_directory(const std::string& dir) {
    std::vector<std::string> files;
    DIR* dp;
    struct dirent* entry;
    if ((dp = opendir(dir.c_str())) != NULL) {
        while ((entry = readdir(dp)) != NULL) {
            std::string filename = entry->d_name;
            if (filename.find(".txt") != std::string::npos) {
                files.push_back(filename);
            }
        }
        closedir(dp);
    } else {
        perror("opendir");
    }
    return files;
}

// 读取 .txt 文件并转换为 std::vector<float>
std::vector<float> read_param(const std::string& filepath) {
    std::vector<float> data;
    std::ifstream file(filepath);
    if (file.is_open()) {
        float value;
        while (file >> value) {
            data.push_back(value);
        }
        file.close();
    } else {
        std::cerr << "Unable to open file: " << filepath << std::endl;
    }
    return data;
}

std::map<std::string, std::vector<float>> read_params(std::string dir) {
    // std::string dir = "."; // 当前目录
    std::map<std::string, std::vector<float>> params;

    // 获取目录中的所有 .txt 文件
    std::vector<std::string> param_files = get_files_in_directory(dir);
    for (const auto& file : param_files) {
        std::string filename = file.substr(0, file.find_last_of(".")); // 获取不带扩展名的文件名
        params[filename] = read_param(dir + "/" + file);
    }

    // // 访问参数时可以使用 params["conv1_weight"]
    // for (const auto& kv : params) {
    //     std::cout << "Key: " << kv.first << ", Values: ";
    //     // for (const auto& value : kv.second) {
    //     //     std::cout << value << " ";
    //     // }
    //     std::cout << std::endl;
    // }

    return params;
}

/****************************************************************************************
 * 读取训练集数据
 ****************************************************************************************/

using namespace H5;
void read_h5_file(const std::string& file_path, std::vector<std::vector<float>>& list_of_points, std::vector<int>& list_of_labels) {
    try {
        // 打开文件
        H5File file(file_path, H5F_ACC_RDONLY);

        // 获取文件中的所有数据集名称
        std::vector<std::string> dataset_names;
        hsize_t num_objs = file.getNumObjs();
        for (hsize_t i = 0; i < num_objs; i++) {
            dataset_names.push_back(file.getObjnameByIdx(i));
        }

        // 读取每个数据集
        for (const auto& name : dataset_names) {
            DataSet dataset = file.openDataSet(name + "/points");
            DataSpace dataspace = dataset.getSpace();

            // 获取数据集的维度
            hsize_t dims[2];
            dataspace.getSimpleExtentDims(dims, NULL);

            // 读取数据
            std::vector<float> points(dims[0] * dims[1]);
            dataset.read(points.data(), PredType::NATIVE_FLOAT);

            // 存储点云数据
            list_of_points.push_back(points);

            // 读取标签
            Attribute label_attr = file.openGroup(name).openAttribute("label");
            int label;
            label_attr.read(PredType::NATIVE_INT, &label);

            // 存储标签
            list_of_labels.push_back(label);
        }
    } catch (FileIException& error) {
        error.printErrorStack();
    } catch (DataSetIException& error) {
        error.printErrorStack();
    } catch (DataSpaceIException& error) {
        error.printErrorStack();
    } catch (DataTypeIException& error) {
        error.printErrorStack();
    }
}

void HANDLE_ERROR(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        // 处理错误，例如退出程序等
    }
}

int main(int argc, char *argv[]) {
    // std::string dir = "parameters";  // 第一个参数是程序所在的目录，这个目录是存放前一步训练模型参数文件的目录，从这个目录下读取模型参数文件，相对于这个目录读取测试集点云数据和标签
    
    std::string dir = argv[1];  // 第一个参数是程序所在的目录，这个目录是存放前一步训练模型参数文件的目录，从这个目录下读取模型参数文件，相对于这个目录读取测试集点云数据和标签
    // cout << dir;
    
    // 读取模型参数
    auto params = read_params(dir);

    std::string file_path = "./data/test_point_clouds.h5";
    std::vector<std::vector<float>> list_of_points;
    std::vector<int> list_of_labels;
    // 读取训练集数据
    read_h5_file(file_path, list_of_points, list_of_labels);

    // 开始计时，使用chrono计时，不支持其它计时方式

    int correct_count = 0;
    int total_count = list_of_points.size();
    // __constant__ float * data;
    // hipMalloc(&data,1000*22500*3*sizeof(float));

    // freopen("out", "w", stdout);

    // 将vector的数据全部转化为cuda的device端的数组，存储转置前和转置后的点云数据
    // float array_of_points[total_count][22500*3];
    // memset(array_of_points, 0, total_count * 22500*3 * sizeof(float));
    // float array_of_points_T[total_count][22500*3];
    // puts("=======================================================");
    //赋值
    float * device_array_1000_22500_3[total_count];
    float * device_array_1000_22500_3_T[total_count];
    for (int i=0;i<total_count;++i){
        vector<float> current_point = list_of_points[i];
        float array_of_points[22500*3];
        int size = current_point.size();
        for (int j=0;j<22500*3;++j){
            if (j<size)
                array_of_points[j] = current_point[j];
            else 
                array_of_points[j] = 0;
        }
        //为转置矩阵赋值
        float array_of_points_T[22500*3];
        int count = 0;
        for (int j=0;j<3;++j)
            for (int k=j;k<22500*3;k+=3){
                array_of_points_T[count ++ ] = array_of_points[k];
            } 
        
        // // 转置前和转置后的点云数据转化为cuda的device端的数组
        float * tmp_1; HANDLE_ERROR( hipMalloc((void**)&tmp_1, 22500*3*sizeof(float)));
        float * tmp_2; HANDLE_ERROR( hipMalloc((void**)&tmp_2, 22500*3*sizeof(float)));
        HANDLE_ERROR( hipMemcpy(tmp_1, array_of_points, 22500*3*sizeof(float), hipMemcpyHostToDevice));
        HANDLE_ERROR( hipMemcpy(tmp_2, array_of_points_T, 22500*3*sizeof(float), hipMemcpyHostToDevice));
        device_array_1000_22500_3[i] = tmp_1;
        device_array_1000_22500_3_T[i] = tmp_2;
    }
    // puts("=======================================================");
    // list_of_points.clear();


    float ** batch_device_array_1000_22500_3;
    float ** batch_device_array_1000_22500_3_T;
    HANDLE_ERROR( hipMalloc((void**)&batch_device_array_1000_22500_3, total_count*sizeof(float*)));
    HANDLE_ERROR( hipMalloc((void**)&batch_device_array_1000_22500_3_T, total_count*sizeof(float*)));
    HANDLE_ERROR( hipMemcpy(batch_device_array_1000_22500_3, device_array_1000_22500_3, total_count*sizeof(float*), hipMemcpyHostToDevice));
    HANDLE_ERROR( hipMemcpy(batch_device_array_1000_22500_3_T, device_array_1000_22500_3_T, total_count*sizeof(float*), hipMemcpyHostToDevice));

    map<string, float*> device_params;
    for (auto& kv : params){
        string name = kv.first;
        vector<float> values = kv.second;
        float * tmp;
        HANDLE_ERROR( hipMalloc((void**)&tmp, values.size()*sizeof(float)));
        HANDLE_ERROR( hipMemcpy(tmp, values.data(), values.size()*sizeof(float), hipMemcpyHostToDevice));
        device_params[name] = tmp;
    }

    const int batch_size = 4;

    float * device_64_22500_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_64_22500_BATCH, batch_size*64*22500*sizeof(float)));

    float * device_64_22500_2_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_64_22500_2_BATCH, batch_size*64*22500*sizeof(float)));

    float * device_22500_64_copy_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_22500_64_copy_BATCH, batch_size*22500*64*sizeof(float)));

    float * device_128_22500_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_128_22500_BATCH, batch_size*128*22500*sizeof(float)));

    float * device_1024_22500_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_1024_22500_BATCH, batch_size*1024*22500*sizeof(float)));

    float * device_1024_1_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_1024_1_BATCH, batch_size*1024*1*sizeof(float)));

    float * device_512_1_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_512_1_BATCH, batch_size*512*1*sizeof(float)));

    float * device_256_1_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_256_1_BATCH, batch_size*256*1*sizeof(float)));

    float * device_9_1_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_9_1_BATCH, batch_size*9*1*sizeof(float)));

    float * device_3_22500_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_3_22500_BATCH, batch_size*3*22500*sizeof(float)));

    float * device_64_64_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_64_64_BATCH, batch_size*64*64*sizeof(float)));

    float * device_4096_1_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_4096_1_BATCH, batch_size*4096*1*sizeof(float)));

    float * device_10_1_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_10_1_BATCH, batch_size*10*1*sizeof(float)));

    float * device_1000_10_1_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_1000_10_1_BATCH, batch_size*1000*10*1*sizeof(float)));

    float * device_max_ans_BATCH;
    HANDLE_ERROR( hipMalloc((void**)&device_max_ans_BATCH, batch_size*1*sizeof(float)));

    float * host_max_ans_BATCH = (float *) malloc(batch_size*1*sizeof(float));

    auto start = std::chrono::high_resolution_clock::now();


    // const int numStreams = 10;
    // hipStream_t streams[numStreams];
    // for (int i = 0; i < numStreams; ++i) {
    //     HANDLE_ERROR(hipStreamCreate(&streams[i])) ;
    // }



    for (int i=0;i<total_count;i+=batch_size){
        // puts("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!");
        int start_index = i;
        int end_index = min(i+batch_size, total_count);
        int current_batch_size = end_index - start_index;

        dim3 threadPerBlock_1 (4, 64);  // 2，16的话 2 会变慢
        dim3 blockPerGrid_1 ((64+threadPerBlock_1.x-1)/threadPerBlock_1.x,(22500+threadPerBlock_1.y-1)/threadPerBlock_1.y, batch_size);

        conv1d_norm_relu_begin<<<blockPerGrid_1,threadPerBlock_1>>>(
            device_64_22500_BATCH,
            64, 3, 22500,
            device_params["feat.stn.conv1.weight"],
            start_index, end_index,
            batch_device_array_1000_22500_3_T,
            device_params["feat.stn.conv1.bias"],
//"feat.stn.bn1.weight","feat.stn.bn1.bias","feat.stn.bn1.running_mean","feat.stn.bn1.running_var"
            device_params["feat.stn.bn1.weight"],
            device_params["feat.stn.bn1.bias"],
            device_params["feat.stn.bn1.running_mean"],
            device_params["feat.stn.bn1.running_var"]
        );

        // print_device_array(device_64_22500_BATCH, 64, 22500, batch_size, true);

        // break;
        dim3 threadPerBlock_2 (4, 16);
        dim3 blockPerGrid_2 ((128+threadPerBlock_2.x-1)/threadPerBlock_2.x,(22500+threadPerBlock_2.y-1)/threadPerBlock_2.y, batch_size);


        conv1d_norm_relu_batch<<<blockPerGrid_2,threadPerBlock_2>>>(
            device_128_22500_BATCH,
            128, 64, 22500,
//"feat.stn.conv2.weight","feat.stn.conv2.bias"
            device_params["feat.stn.conv2.weight"],
            // start_index, end_index,
            current_batch_size,
            device_64_22500_BATCH,
            device_params["feat.stn.conv2.bias"],
// "feat.stn.bn2.weight", "feat.stn.bn2.bias", "feat.stn.bn2.running_mean", "feat.stn.bn2.running_var"
            device_params["feat.stn.bn2.weight"],
            device_params["feat.stn.bn2.bias"],
            device_params["feat.stn.bn2.running_mean"],
            device_params["feat.stn.bn2.running_var"]
        );

        // print_device_array(device_128_22500_BATCH, 128*batch_size, 22500, true);

        // break;

        dim3 threadPerBlock_3 (4, 64);
        dim3 blockPerGrid_3 ((1024+threadPerBlock_3.x-1)/threadPerBlock_3.x,(22500+threadPerBlock_3.y-1)/threadPerBlock_3.y, batch_size);


        conv1d_norm_relu_batch<<<blockPerGrid_3,threadPerBlock_3>>>(
            device_1024_22500_BATCH,
            1024, 128, 22500,
//"feat.stn.conv3.weight","feat.stn.conv3.bias"
            device_params["feat.stn.conv3.weight"],
            // start_index, end_index,
            current_batch_size,
            device_128_22500_BATCH,
            device_params["feat.stn.conv3.bias"],
// "feat.stn.bn3.weight", "feat.stn.bn3.bias", "feat.stn.bn3.running_mean", "feat.stn.bn3.running_var"
            device_params["feat.stn.bn3.weight"],
            device_params["feat.stn.bn3.bias"],
            device_params["feat.stn.bn3.running_mean"],
            device_params["feat.stn.bn3.running_var"]
        );

        // print_device_array(device_1024_22500_BATCH, 1024*batch_size, 22500, true);


        // max_matrix_batch<<<(dim3)(1024, batch_size),16>>>(
        dim3 blocktmpmax (1024, batch_size);
        max_matrix_batch<<<blocktmpmax,4>>>(
            device_1024_1_BATCH, 
            current_batch_size,
            device_1024_22500_BATCH, 1024, 22500);


        // print_device_array(device_1024_1_BATCH, 1024*batch_size, 1, false);

        // break;

        dim3 threadPerBlock_4 (4, 4);
        dim3 blockPerGrid_4 ((512+threadPerBlock_4.x-1)/threadPerBlock_4.x,(1+threadPerBlock_4.y-1)/threadPerBlock_4.y, batch_size);


        conv1d_norm_relu_batch<<<blockPerGrid_4,threadPerBlock_4>>>(
            device_512_1_BATCH,
            512, 1024, 1,
//"feat.stn.fc1.weight","feat.stn.fc1.bias"
            device_params["feat.stn.fc1.weight"],
            // start_index, end_index,
            current_batch_size,
            device_1024_1_BATCH,
            device_params["feat.stn.fc1.bias"],
// "feat.stn.bn4.weight", "feat.stn.bn4.bias", "feat.stn.bn4.running_mean", "feat.stn.bn4.running_var"
            device_params["feat.stn.bn4.weight"],
            device_params["feat.stn.bn4.bias"],
            device_params["feat.stn.bn4.running_mean"],
            device_params["feat.stn.bn4.running_var"]
        );

        dim3 threadPerBlock_5 (4, 4);
        dim3 blockPerGrid_5 ((256+threadPerBlock_5.x-1)/threadPerBlock_5.x,(1+threadPerBlock_5.y-1)/threadPerBlock_5.y, batch_size);


        conv1d_norm_relu_batch<<<blockPerGrid_5,threadPerBlock_5>>>(
            device_256_1_BATCH,
            256, 512, 1,
//"feat.stn.fc2.weight","feat.stn.fc2.bias"
            device_params["feat.stn.fc2.weight"],
            // start_index, end_index,
            current_batch_size,
            device_512_1_BATCH,
            device_params["feat.stn.fc2.bias"],
// "feat.stn.bn5.weight", "feat.stn.bn5.bias", "feat.stn.bn5.running_mean", "feat.stn.bn5.running_var"
            device_params["feat.stn.bn5.weight"],
            device_params["feat.stn.bn5.bias"],
            device_params["feat.stn.bn5.running_mean"],
            device_params["feat.stn.bn5.running_var"]
        );


        dim3 blocktmp9 (9, batch_size);
        conv1d_9_batch<<<blocktmp9,1>>>(
            device_9_1_BATCH, 
            device_params["feat.stn.fc3.weight"], 
            current_batch_size,
            device_256_1_BATCH, 
            device_params["feat.stn.fc3.bias"]);

//------------------------正确-----------------------------------------------
        // print_device_array(device_9_1_BATCH, 9, 1, batch_size, false);
        // break;

        dim3 threadPerBlock_6 (16, 4);
        dim3 blockPerGrid_6 ((22500+threadPerBlock_6.x-1)/threadPerBlock_6.x,(3+threadPerBlock_6.y-1)/threadPerBlock_6.y, batch_size);
        matrix_array_9_batch<<<blockPerGrid_6,threadPerBlock_6>>>(
            start_index, end_index,
            device_3_22500_BATCH , 
            batch_device_array_1000_22500_3, 
            device_9_1_BATCH, 
            22500, 3, 3);

        // print_device_array(device_3_22500_BATCH, 3, 22500, batch_size, true);

        // break;

        dim3 threadPerBlock_7 (4, 16);
        dim3 blockPerGrid_7 ((64+threadPerBlock_7.x-1)/threadPerBlock_7.x,(22500+threadPerBlock_7.y-1)/threadPerBlock_7.y, batch_size);
        conv1d_norm_relu_copy_T_batch<<<blockPerGrid_7,threadPerBlock_7>>>(
            current_batch_size,
            device_64_22500_BATCH,
            device_22500_64_copy_BATCH,
            64, 3, 22500,
//"feat.conv1.weight","feat.conv1.bias"
            device_params["feat.conv1.weight"],
            device_3_22500_BATCH,
            device_params["feat.conv1.bias"],
// "feat.bn1.weight", "feat.bn1.bias", "feat.bn1.running_mean", "feat.bn1.running_var"
            device_params["feat.bn1.weight"],
            device_params["feat.bn1.bias"],
            device_params["feat.bn1.running_mean"],
            device_params["feat.bn1.running_var"]
        );

        // print_device_array(device_64_22500_BATCH, 64*batch_size, 22500, true);

        // break;

        dim3 threadPerBlock_8 (4, 16);
        dim3 blockPerGrid_8 ((64+threadPerBlock_8.x-1)/threadPerBlock_8.x,(22500+threadPerBlock_8.y-1)/threadPerBlock_8.y, batch_size);

        conv1d_norm_relu_batch<<<blockPerGrid_8,threadPerBlock_8>>>(
            device_64_22500_2_BATCH,
            64, 64, 22500,
// "feat.fstn.conv1.weight","feat.fstn.conv1.bias"
            device_params["feat.fstn.conv1.weight"],
            current_batch_size,
            device_64_22500_BATCH,
            device_params["feat.fstn.conv1.bias"],
// "feat.fstn.bn1.weight","feat.fstn.bn1.bias","feat.fstn.bn1.running_mean","feat.fstn.bn1.running_var"
            device_params["feat.fstn.bn1.weight"],
            device_params["feat.fstn.bn1.bias"],
            device_params["feat.fstn.bn1.running_mean"],
            device_params["feat.fstn.bn1.running_var"]
        );
        dim3 threadPerBlock_9 (4, 16);
        dim3 blockPerGrid_9 ((128+threadPerBlock_9.x-1)/threadPerBlock_9.x,(22500+threadPerBlock_9.y-1)/threadPerBlock_9.y, batch_size);
        conv1d_norm_relu_batch<<<blockPerGrid_9,threadPerBlock_9>>>(
            device_128_22500_BATCH,
            128, 64, 22500,
// "feat.fstn.conv2.weight","feat.fstn.conv2.bias"
            device_params["feat.fstn.conv2.weight"],
            current_batch_size,
            device_64_22500_2_BATCH,
            device_params["feat.fstn.conv2.bias"],
// "feat.fstn.bn2.weight","feat.fstn.bn2.bias","feat.fstn.bn2.running_mean","feat.fstn.bn2.running_var"
            device_params["feat.fstn.bn2.weight"],
            device_params["feat.fstn.bn2.bias"],
            device_params["feat.fstn.bn2.running_mean"],
            device_params["feat.fstn.bn2.running_var"]
        );
        dim3 threadPerBlock_10 (4, 64); // 这里不能放6 64 会变慢
        dim3 blockPerGrid_10 ((1024+threadPerBlock_10.x-1)/threadPerBlock_10.x,(22500+threadPerBlock_10.y-1)/threadPerBlock_10.y, batch_size);
        conv1d_norm_relu_batch<<<blockPerGrid_10,threadPerBlock_10>>>(
            device_1024_22500_BATCH,
            1024, 128, 22500,
// "feat.fstn.conv3.weight","feat.fstn.conv3.bias"
            device_params["feat.fstn.conv3.weight"],
            current_batch_size,
            device_128_22500_BATCH,
            device_params["feat.fstn.conv3.bias"],
// "feat.fstn.bn3.weight","feat.fstn.bn3.bias","feat.fstn.bn3.running_mean","feat.fstn.bn3.running_var"
            device_params["feat.fstn.bn3.weight"],
            device_params["feat.fstn.bn3.bias"],
            device_params["feat.fstn.bn3.running_mean"],
            device_params["feat.fstn.bn3.running_var"]
        );
        max_matrix_batch<<<blocktmpmax,16>>>(
            device_1024_1_BATCH, current_batch_size, device_1024_22500_BATCH, 1024, 22500);
        dim3 threadPerBlock_11 (4, 4);
        dim3 blockPerGrid_11 ((512+threadPerBlock_11.x-1)/threadPerBlock_11.x,(1+threadPerBlock_11.y-1)/threadPerBlock_11.y, batch_size);
        conv1d_norm_relu_batch<<<blockPerGrid_11,threadPerBlock_11>>>(
            device_512_1_BATCH,
            512, 1024, 1,
// "feat.fstn.fc1.weight","feat.fstn.fc1.bias"
            device_params["feat.fstn.fc1.weight"],
            current_batch_size,
            device_1024_1_BATCH,
            device_params["feat.fstn.fc1.bias"],
// "feat.fstn.bn4.weight","feat.fstn.bn4.bias","feat.fstn.bn4.running_mean","feat.fstn.bn4.running_var"
            device_params["feat.fstn.bn4.weight"],
            device_params["feat.fstn.bn4.bias"],
            device_params["feat.fstn.bn4.running_mean"],
            device_params["feat.fstn.bn4.running_var"]
        );
        dim3 threadPerBlock_12 (4, 4);
        dim3 blockPerGrid_12 ((256+threadPerBlock_12.x-1)/threadPerBlock_12.x,(1+threadPerBlock_12.y-1)/threadPerBlock_12.y, batch_size);
        conv1d_norm_relu_batch<<<blockPerGrid_12,threadPerBlock_12>>>(
            device_256_1_BATCH,
            256, 512, 1,
// "feat.fstn.fc2.weight","feat.fstn.fc2.bias"
            device_params["feat.fstn.fc2.weight"],
            current_batch_size,
            device_512_1_BATCH,
            device_params["feat.fstn.fc2.bias"],
// "feat.fstn.bn5.weight","feat.fstn.bn5.bias","feat.fstn.bn5.running_mean","feat.fstn.bn5.running_var"
            device_params["feat.fstn.bn5.weight"],
            device_params["feat.fstn.bn5.bias"],
            device_params["feat.fstn.bn5.running_mean"],
            device_params["feat.fstn.bn5.running_var"]
        );
        // print_device_array(device_256_1_BATCH, 256*batch_size, 1, false);
        dim3 blocktmpadd64 (4096, batch_size);
        conv1d_4096_add_64_64_batch<<<blocktmpadd64,1>>>(
            current_batch_size,
            device_4096_1_BATCH,
            4096, 256, 1,
// "feat.fstn.fc3.weight","feat.fstn.fc3.bias"
            device_params["feat.fstn.fc3.weight"],
            device_256_1_BATCH,
            device_params["feat.fstn.fc3.bias"]
        );
        // print_device_array(device_4096_1_BATCH, 4096*batch_size, 1, false);
        // break;
        dim3 threadPerBlock_13 (16, 4);
        dim3 blockPerGrid_13 ((22500+threadPerBlock_13.x-1)/threadPerBlock_13.x,(64+threadPerBlock_13.y-1)/threadPerBlock_13.y, batch_size);
        conv1d_22500_64_64_trans_result_batch<<<blockPerGrid_13,threadPerBlock_13>>>(
            current_batch_size,
            device_64_22500_BATCH, // transed_result
            22500, 64, 64,
            device_22500_64_copy_BATCH,
            device_4096_1_BATCH
        );
        // print_device_array(device_64_22500_BATCH, 64, 22500, batch_size,true);
        // break;
        dim3 threadPerBlock_14 (4, 16);
        dim3 blockPerGrid_14 ((128+threadPerBlock_14.x-1)/threadPerBlock_14.x,(22500+threadPerBlock_14.y-1)/threadPerBlock_14.y, batch_size);
        conv1d_norm_relu_batch<<<blockPerGrid_14,threadPerBlock_14>>>(
            device_128_22500_BATCH,
            128, 64, 22500,
// "feat.conv2.weight","feat.conv2.bias"
            device_params["feat.conv2.weight"],
            current_batch_size,
            device_64_22500_BATCH,
            device_params["feat.conv2.bias"],
// "feat.bn2.weight","feat.bn2.bias","feat.bn2.running_mean","feat.bn2.running_var"
            device_params["feat.bn2.weight"],
            device_params["feat.bn2.bias"],
            device_params["feat.bn2.running_mean"],
            device_params["feat.bn2.running_var"]
        );
        dim3 threadPerBlock_15 (4, 64);
        dim3 blockPerGrid_15 ((1024+threadPerBlock_15.x-1)/threadPerBlock_15.x,(22500+threadPerBlock_15.y-1)/threadPerBlock_15.y, batch_size);
        conv1d_norm_batch<<<blockPerGrid_15,threadPerBlock_15>>>(
            current_batch_size,
            device_1024_22500_BATCH,
            1024, 128, 22500,
// "feat.conv3.weight","feat.conv3.bias"
            device_params["feat.conv3.weight"],
            device_128_22500_BATCH,
            device_params["feat.conv3.bias"],
// "feat.bn3.weight","feat.bn3.bias","feat.bn3.running_mean","feat.bn3.running_var"
            device_params["feat.bn3.weight"],
            device_params["feat.bn3.bias"],
            device_params["feat.bn3.running_mean"],
            device_params["feat.bn3.running_var"]
        );
        max_matrix_batch<<<blocktmpmax,1>>>(
            device_1024_1_BATCH, 
            current_batch_size,
            device_1024_22500_BATCH, 1024, 22500);
        // print_device_array(device_1024_1_BATCH, 1024, 1,batch_size, false);
        // break;
        dim3 threadPerBlock_16 (4, 4);
        dim3 blockPerGrid_16 ((512+threadPerBlock_16.x-1)/threadPerBlock_16.x,(1+threadPerBlock_16.y-1)/threadPerBlock_16.y, batch_size);
        conv1d_norm_relu_batch<<<blockPerGrid_16,threadPerBlock_16>>>(
            device_512_1_BATCH,
            512, 1024, 1,
// "fc1.weight","fc1.bias"
            device_params["fc1.weight"],
            current_batch_size,
            device_1024_1_BATCH,
            device_params["fc1.bias"],
// "bn1.weight","bn1.bias","bn1.running_mean","bn1.running_var"
            device_params["bn1.weight"],
            device_params["bn1.bias"],
            device_params["bn1.running_mean"],
            device_params["bn1.running_var"]
        );
        dim3 threadPerBlock_17 (4, 4);
        dim3 blockPerGrid_17 ((256+threadPerBlock_17.x-1)/threadPerBlock_17.x,(1+threadPerBlock_17.y-1)/threadPerBlock_17.y, batch_size);
        conv1d_norm_relu_batch<<<blockPerGrid_17,threadPerBlock_17>>>(
            device_256_1_BATCH,
            256, 512, 1,
// "fc2.weight","fc2.bias"
            device_params["fc2.weight"],
            current_batch_size,
            device_512_1_BATCH,
            device_params["fc2.bias"],
// "bn2.weight","bn2.bias","bn2.running_mean","bn2.running_var"
            device_params["bn2.weight"],
            device_params["bn2.bias"],
            device_params["bn2.running_mean"],
            device_params["bn2.running_var"]
        );
        // print_device_array(device_256_1_BATCH, 256, 1,batch_size, false);
        // break;
        dim3 blocktmpans (10, batch_size);
        conv1d_log_softmax_ans_batch<<<blocktmpans,1>>>(
            current_batch_size,
            device_10_1_BATCH,
            10, 256,    //b_col = 1
// "fc3.weight","fc3.bias"
            device_params["fc3.weight"],
            device_256_1_BATCH,
            device_params["fc3.bias"]
        );
        // print_device_array(device_10_1_BATCH, 10, 1,batch_size, false);
        // if (i==2)
        // break;
        get_max_ans_batch<<<batch_size,1>>>(
            current_batch_size,
            device_10_1_BATCH,
            device_max_ans_BATCH
            );
        // print_device_array(device_max_ans_BATCH, 1, 1,batch_size, false);
        hipMemcpyAsync(
            host_max_ans_BATCH,
            device_max_ans_BATCH,
            sizeof(float) * current_batch_size,
            hipMemcpyDeviceToHost
        );

        for (int j = 0; j < current_batch_size; j++) {
            // print_float(host_max_ans_BATCH[j]);
            if (host_max_ans_BATCH[j] == list_of_labels[j + start_index]) {
                correct_count++;
            }
        }
        // break;
    }

    // 向主机端同步以等待所有异步调用的GPU kernel执行完毕，这句必须要有
    hipDeviceSynchronize();

    // 结束计时
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;

    // 输出结果，请严格保持此输出格式，并把0.0001替换成实际的准确率，请不要输出除了此结果之外的任何内容！！！
    std::cout << std::fixed << std::setprecision(4) << diff.count() << ":" <<(float)correct_count / (float)total_count;

    return 0;
}


__global__ void conv1d_norm_relu_begin(float * output,  int a_row, int a_col, int b_col, float * weight, int start_index, int end_index, float ** input, float * bias, float * bn_weight, float * bn_bias, float * bn_running_mean, float * bn_running_var){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idx*b_col+idy;

    int outputid = id + blockIdx.z * a_row * b_col;

    if (start_index + blockIdx.z < end_index && idx<a_row && idy<b_col){
        float sum = 0;
        for (int i=0;i<a_col;i++){
            sum += weight[idx*a_col+i] * input[start_index + blockIdx.z][i*b_col+idy];
        }
        sum += bias[idx];
        float x_mean = bn_running_mean[idx];
        float x_var = bn_running_var[idx];
        float bn1_w_val = bn_weight[idx];
        float bn1_b_val = bn_bias[idx];

        float norm_res = (sum - x_mean) / sqrt(x_var + (1e-5));
        float ans = norm_res * bn1_w_val + bn1_b_val;
        ans = ans > 0 ? ans : 0;
        output[outputid] = ans;
        // output[outputid] = sum;
        // output[id] = sum;
    }
}


__global__ void conv1d_norm_relu_batch(float * output,  int a_row, int a_col, int b_col, float * weight, int current_batch_size,float * input, float * bias, float * bn_weight, float * bn_bias, float * bn_running_mean, float * bn_running_var){

    int batch_index = blockIdx.z;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idx*b_col+idy;

    int outputid = id + blockIdx.z * a_row * b_col;

    if (batch_index < current_batch_size && idx<a_row && idy<b_col){
        float sum = 0;
        for (int i=0;i<a_col;i++){
            int inputid = i*b_col+idy + batch_index * a_col * b_col;
            sum += weight[idx*a_col+i] * input[inputid];
        }
        sum += bias[idx];
        float x_mean = bn_running_mean[idx];
        float x_var = bn_running_var[idx];
        float bn1_w_val = bn_weight[idx];
        float bn1_b_val = bn_bias[idx];

        float norm_res = (sum - x_mean) / sqrt(x_var + (1e-5));
        float ans = norm_res * bn1_w_val + bn1_b_val;
        ans = ans > 0 ? ans : 0;
        output[outputid] = ans;
        // output[outputid] = sum;
        // output[id] = sum;
    }
}

// __global__ void conv1d_norm_relu(float * output,  int a_row, int a_col, int b_col, float * weight, float ** input, float * bias, float * bn_weight, float * bn_bias, float * bn_running_mean, float * bn_running_var){
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int id = idx*b_col+idy;
//     if (idx<a_row && idy<b_col){
//         float sum = 0;
//         for (int i=0;i<a_col;i++){
//             sum += weight[idx*a_col+i] * input[0][i*b_col+idy];
//         }
//         sum += bias[idx];
//         float x_mean = bn_running_mean[idx];
//         float x_var = bn_running_var[idx];
//         float bn1_w_val = bn_weight[idx];
//         float bn1_b_val = bn_bias[idx];

//         float norm_res = (sum - x_mean) / sqrt(x_var + (1e-5));
//         float ans = norm_res * bn1_w_val + bn1_b_val;
//         ans = ans > 0 ? ans : 0;
//         output[id] = ans;
//         // output[id] = sum;
//     }
// }

__global__ void conv1d_log_softmax_ans_batch(int current_batch_size, float * output, int a_row, int a_col, float * weight, float * input, float * bias){
    int batch_index = blockIdx.y;
    int idx = blockIdx.x;
    if (batch_index < current_batch_size && idx<a_row){
        float num = 0;
        for (int i=0;i<a_col;i++){
            int inputid = i + batch_index * a_col;
            num += weight[idx*a_col+i] * input[inputid];
        }
        num += bias[idx];
        output[idx + batch_index * a_row] = num;
        }
}

__global__ void get_max_ans_batch(
            int current_batch_size,
            float * device_10_1_BATCH,
            float *device_max_ans_BATCH
            ){
    int max_idx = 0;
    float max_val = device_10_1_BATCH[blockIdx.x * 10];
    for (int i=1;i<10;++i){
        float current_val = device_10_1_BATCH[blockIdx.x * 10 + i];
        if (current_val > max_val){
            max_val = current_val;
            max_idx = i;
        }
    }
    device_max_ans_BATCH[blockIdx.x] = max_idx;
    // __shared__ float max_val[10];
    // __shared__ int max_idx[10];
    // int batch_index = blockIdx.x;
    // int idx = threadIdx.x;
    // if (batch_index < current_batch_size && idx<10){
    //     if (idx == 0){
    //         max_val[batch_index] = device_10_1_BATCH[batch_index * 10];
    //         // max_val = device_10_1_BATCH[];
    //         max_idx[batch_index] = 0;
    //     }
    //     else{
    //         float current_val = device_10_1_BATCH[batch_index * 10 + idx];
    //         if (current_val > max_val[idx]){
    //             max_val[batch_index] = current_val;
    //             max_idx[batch_index] = idx;
    //         }
    //     }
    // }
    // __syncthreads();
    // device_max_ans_BATCH[batch_index] = max_idx[batch_index];
}

void print_device_float(float * num){
    float * host_num = (float *)malloc(sizeof(float));
    hipMemcpy(host_num, num, sizeof(float), hipMemcpyDeviceToHost);
    std::cout << *host_num << std::endl;
    free(host_num);
}

__global__ void conv1d_norm_batch(
    int current_batch_size,
    float * output,  int a_row, int a_col, int b_col, float * weight, float * input, float * bias, float * bn_weight, float * bn_bias, float * bn_running_mean, float * bn_running_var){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idx*b_col+idy;
    int batch_index = blockIdx.z;
    if (idx<a_row && idy<b_col){
        float sum = 0;
        for (int i=0;i<a_col;i++){
            int inputid = i*b_col+idy + batch_index * a_col * b_col;
            sum += weight[idx*a_col+i] * input[inputid];
        }
        sum += bias[idx];
        float x_mean = bn_running_mean[idx];
        float x_var = bn_running_var[idx];
        float bn1_w_val = bn_weight[idx];
        float bn1_b_val = bn_bias[idx];

        float norm_res = (sum - x_mean) / sqrt(x_var + (1e-5));
        output[id + batch_index * a_row * b_col] = norm_res * bn1_w_val + bn1_b_val;
        // output[id] = sum;
    }
}

__global__ void conv1d_4096_add_64_64_batch(
    int current_batch_size,
    float * output,  int a_row, 
    int a_col, int b_col, float * weight, float * input, float * bias){
    //a_row = 4096
    //a_col = 256
    //b_col = 1
    // __shared__ float * array_256_1 = input;

    int idx = blockIdx.x;
    int batch_index = blockIdx.y;
    //  256 * 1
    if (batch_index< current_batch_size && idx<a_row){
        float sum = 0;
        for (int i=0;i<a_col;i++){
            int inputid = i + batch_index * a_col * b_col;
            sum += weight[idx*a_col+i] * input[inputid];
        }
        sum += bias[idx];
        output[idx + batch_index * a_row * b_col] = sum;
        int current_row = idx / 64;
        if (idx == current_row + current_row * 64)
            output[idx + batch_index * a_row * b_col] ++;
    }
}

__global__ void conv1d_22500_64_64_trans_result_batch(
    int current_batch_size,
    float * output, int a_row, int a_col, int b_col, 
    float * input_a, float * input_b)
{
    int batch_index = blockIdx.z;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (batch_index < current_batch_size &&idx<a_row && idy<a_col){
        float sum = 0;
        for (int i=0;i<a_col;i++){
            int inputa_id = idx*a_col+i + batch_index * a_row * a_col;
            int inputb_id = i*b_col+idy + batch_index * a_col * b_col;
            sum += input_a[inputa_id] * input_b[inputb_id];
        }
        output[idy*a_row+idx + batch_index * a_row * b_col] = sum;
    }
}

__global__ void conv1d_norm_relu_copy_T_batch(
    int current_batch_size,
    float * output,  float * output_T, int a_row, int a_col, int b_col, float * weight, float * input, float * bias, float * bn_weight, float * bn_bias, float * bn_running_mean, float * bn_running_var){


    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int batch_index = blockIdx.z;

    if (batch_index< current_batch_size && idx<a_row && idy<b_col){
        int id = idx*b_col+idy;
        int id_T = idy*a_row+idx;
        float sum = 0;
        for (int i=0;i<a_col;i++){
            int inputid = i*b_col+idy + batch_index * a_col * b_col;
            sum += weight[idx*a_col+i] * input[inputid];
        }
        sum += bias[idx];
        float x_mean = bn_running_mean[idx];
        float x_var = bn_running_var[idx];
        float bn1_w_val = bn_weight[idx];
        float bn1_b_val = bn_bias[idx];

        float norm_res = (sum - x_mean) / sqrt(x_var + (1e-5));
        float ans = norm_res * bn1_w_val + bn1_b_val;
        ans = ans > 0 ? ans : 0;
        output[id + batch_index * a_row * b_col] = ans;
        output_T[id_T + batch_index * a_row * b_col] = ans;
        // output[id] = sum;
    }
}

__global__ void max_matrix_batch(float * output, int current_batch_size, float * input, int row, int col){

    // const int shared_memory_size = 20;

    int batch_index = blockIdx.y;


    int id = blockIdx.x;

    int tid = threadIdx.x;

    int thread_num = blockDim.x;

    int output_index = id + batch_index * row;

    __shared__ float max_val;
    if (id<row){
        // float max_val = input[id*col + batch_index * row * col];
        // for (int i=1;i<col;i++){
        //     int inputid = i + id * col + batch_index * row * col;
            
        //     if (max_val < input[inputid]){
        //         max_val = input[inputid];
        //     }
        // }
        // output[output_index] = max_val;

        int each_thread_col = col / thread_num;
        int start_col = each_thread_col * tid;
        int end_col = start_col + each_thread_col;
        for (int i=start_col;i<end_col;i++){
            int inputid = i + id * col + batch_index * row * col;
            float val = input[inputid];
            if (i==0){
                max_val = val;
            }
            else if (max_val < val){
                max_val = val;
            }
        }
        __syncthreads();
        output[output_index] = max_val;
    }
}

__global__ void conv1d_9_batch(float * output, float * weight, 
int current_batch_size, float * input, float * bias){

    int idx = blockIdx.x ;

    int batch_index = blockIdx.y ;

    int output_index = idx + batch_index * 9;

    if (batch_index < current_batch_size && idx<9){
        float sum = 0;
        for (int i=0;i<256;i++){
            int inputid = i + batch_index * 256;
            sum += weight[idx*256+i] * input[inputid];
        }
        sum += bias[idx];
        output[output_index] = sum;
        if (idx==0 || idx==4 || idx==8)
            output[output_index] += 1;
    }
}


__global__ void matrix_array_9_batch(
    int start_index, int end_index,
    float * output, float ** a, float * b, int a_row, int a_col, int b_col){

    int batch_index = start_index + blockIdx.z;
//a: 22500 * 3
//b: 3 * 3
//output: 22500 * 3 * 3

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (batch_index < end_index && idx<a_row && idy<b_col){
        // int id = idx*a_col+idy;
        int id_T = idy*a_row+idx;
        float sum = 0;
        for (int i=0;i<a_col;i++){
            int b_id = i*b_col+idy + blockIdx.z * a_col * b_col;
            sum += a[batch_index][idx*a_col+i] * b[b_id];
        }
        output[id_T + blockIdx.z * a_row * b_col] = sum;
    }

}

__global__ void matrix_array_9(float * output, float * a, float * b, int a_row, int a_col, int b_col){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx<a_row && idy<b_col){
        // int id = idx*a_col+idy;
        int id_T = idy*a_row+idx;
        float sum = 0;
        for (int i=0;i<a_col;i++){
            sum += a[idx*a_col+i] * b[i*b_col+idy];
        }
        output[id_T] = sum;
    }

}

int log_softmax(float * result){

    int max_index = 0;
    float max_value = result[0];
    for (int i=1;i<10;++i){
        if (max_value < result[i]){
            max_value = result[i];
            max_index = i;
        }
    }
    return max_index;
}

void print_device_array( float * array, int row, int col, int batch, bool from22400){
    int length = row * col * batch;
    float * print_array = (float *)malloc (sizeof (float) * length);
    hipMemcpy(print_array, array, sizeof (float) * length, hipMemcpyDeviceToHost);

    // float * print_array = createHostArrayFromDeviceArray(array, sizeof (float) * row * col);
    puts("===============================Check: 0-9 ===========================");

    for (int i=0;i<10;++i){
            printf("%.8f ", print_array[i]);
    }
    puts("");
    puts("===============================Check: 0-9 ===========================");

        for (int i=0;i<10;++i){
            printf("%.8f ", print_array[row*col + i]);
    }
    puts("");
    puts("================================Check: 22400 - 22410=================");
    if (from22400)
        // forloop(i, 0, length)
        forloop(i, 22400, 22410)
            printf("%.8f ", print_array[i]);

    puts("================================End====================================");
    // puts("================================Check: 22400 - 22410=================");
    // if (from22400)
    //     // forloop(i, 0, length)
    //     forloop(i, 22400, 22410)
    //         printf("%.8f ", print_array[64*22500 + i]);

    // puts("================================End====================================");
    free(print_array);
}


void print_device_array_64( float * array, int row, int col, bool from22400){
    int length = row * col;
    float * print_array = (float *)malloc (sizeof (float) * length);
    hipMemcpy(print_array, array, sizeof (float) * length, hipMemcpyDeviceToHost);

    // float * print_array = createHostArrayFromDeviceArray(array, sizeof (float) * row * col);
    puts("===============================Check: 0-9 ===========================");

    for (int i=0;i<64;++i){
        for (int j=0;j<64;++j)
            printf("%.5f ", print_array[i*64+j]);
        puts("");
    }


    
    puts("================================Check: 22400 - 22410=================");
    if (from22400)
        forloop(i, 22400, 22410)
            print_float(print_array[i]);
    puts("================================End====================================");
    free(print_array);
}